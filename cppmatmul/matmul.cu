#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <ostream>
#include <pybind11/buffer_info.h>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <sstream>
#include <stdexcept>

template <typename T>
__global__ void vec_to_scalar(T *vec, T scalar, int num_elements) {
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_elements) {
    vec[idx] = vec[idx] * scalar;
  }
}

template <typename T>
void exec_vec_to_scalar(T *vec, T scalar, int num_elements) {
  dim3 dimBlock(256, 1, 1);
  dim3 dimGrid(ceil((T)num_elements) / dimBlock.x);

  vec_to_scalar<T><<<dimGrid, dimBlock>>>(vec, scalar, num_elements);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    std::stringstream str_stream;
    str_stream << "run_kernel lauch failed " << std::endl;
    str_stream << "dimBlock " << dimBlock.x << ", " << dimBlock.y << std::endl;
    str_stream << "dimGrid " << dimGrid.x << ", " << dimGrid.y << std::endl;
    str_stream << hipGetErrorString(error);
    throw str_stream.str();
  }
}

template <typename T>
void map_vec_to_scalar(pybind11::array_t<T> vec, T scalar) {
  pybind11::buffer_info info = vec.request();

  if (info.ndim != 1) {
    std::stringstream stream;
    stream << "info.ndim != 1" << std::endl;
    stream << "infi.ndim: " << info.ndim << std::endl;
    throw std::runtime_error(stream.str());
  }

  int size = info.shape[0];
  int size_bytes = size * sizeof(T);
  T *gpu_ptr;
  hipError_t error = hipMalloc(&gpu_ptr, size_bytes);

  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }

  T *ptr = reinterpret_cast<T *>(info.ptr);
  error = hipMemcpy(gpu_ptr, ptr, size_bytes, hipMemcpyDeviceToHost);

  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }

  exec_vec_to_scalar<T>(gpu_ptr, scalar, size);

  error = hipMemcpy(ptr, gpu_ptr, size_bytes, hipMemcpyDeviceToHost);
  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }

  error = hipFree(gpu_ptr);
  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }
}

PYBIND11_MODULE(matmul_cuda, matmul) {
  matmul.def("vec_to_scalar", map_vec_to_scalar<double>);
}
